#ifndef _GNU_SOURCE
  #define _GNU_SOURCE
#endif

#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <unistd.h>
#include <numa.h>
#include <numaif.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <inttypes.h>
#include <sys/mman.h>

#define gettime(t) clock_gettime(CLOCK_MONOTONIC_RAW, t)
#define get_sub_seconde(t) (1e-9*(double)t.tv_nsec)
/** return time in second
*/
double get_elapsedtime(void)
{
  struct timespec st;
  int err = gettime(&st);
  if (err !=0) return 0;
  return (double)st.tv_sec + get_sub_seconde(st);
}

//#define N 1E8

#define handle_error_en(en, msg) \
  do { errno = en; perror(msg); exit(EXIT_FAILURE); } while (0)

int main(int argc, char *argv[])
{
  int nb_test = 25;
  int s, j;
  int cpu = -1;
  uint64_t size_in_mbytes = 100;
  bool verbose = false;

  int opt;
  while ((opt = getopt(argc, argv, "vhs:i:")) != -1)
  {
    switch (opt)
    {
      case 's':
        size_in_mbytes = (uint64_t)atoi(optarg);
        break;
      case 'i':
        nb_test = (int)atoi(optarg);
        break;
      case 'v':
        verbose = true;
        break;
      case 'h':
        goto usage;
        break;
      default:
        goto usage;
    }
  }

  if (optind != argc)
  {
usage:
    fprintf(stdout, "CUDA Bench - Async. Memory Transfers Throughput evaluation with NUMA consideration 1.0.0\n");
    fprintf(stdout, "usage: numa_memcpy-async.exe\n\t[-s size in MB]\n\t[-h print this help]\n");
    fprintf(stdout, "\nPlot results using python3:\n");
    fprintf(stdout, "numa_memcpy-async.exe -s <arg> && python3 plot.py <arg>\n");
    exit(EXIT_SUCCESS);
  }

  nb_test+=1;
  cpu_set_t cpuset;
  pthread_t thread;

  thread = pthread_self();

  int numcores = sysconf(_SC_NPROCESSORS_ONLN); // divided by 2 because of hyperthreading
  int numanodes = numa_num_configured_nodes();

  int gpucount = -1;
  hipGetDeviceCount(&gpucount);

  double duration = 0.;
  int *tgpu = (int*)malloc(sizeof(int) * numcores * gpucount);
  double *HtD = (double*)malloc(sizeof(double) * numcores * gpucount);
  double *DtH = (double*)malloc(sizeof(double) * numcores * gpucount);
  double *HtD_gbs = (double*)malloc(sizeof(double) * numcores * gpucount);
  double *DtH_gbs = (double*)malloc(sizeof(double) * numcores * gpucount);
  memset(tgpu, -1, sizeof(int) * numcores * gpucount);
  memset(HtD, 0, sizeof(double) * numcores * gpucount);
  memset(DtH, 0, sizeof(double) * numcores * gpucount);
  memset(HtD_gbs, 0, sizeof(double) * numcores * gpucount);
  memset(DtH_gbs, 0, sizeof(double) * numcores * gpucount);

  double size_in_kbytes = size_in_mbytes*1000;
  double size_in_bytes = size_in_kbytes*1000;

  if(verbose)
  {
#ifdef DEBUG
    fprintf(stdout, "Size of array: %lu Bytes\n", (uint64_t)(size_in_bytes));
    fprintf(stdout, "Size of array: %.2f KB\n", (double)(size_in_kbytes));
#endif
    fprintf(stdout, "Size of array: %.2f MB\n", (double)(size_in_mbytes));

#ifdef DISPLAY_BITS
    float size_kb = (float)(size_in_kbytes * CHAR_BIT);
    float size_mb = (float)(size_in_mbytes * CHAR_BIT);
    fprintf(stdout, "Size of array: %lu bits\n", (uint64_t)(size_in_bytes * CHAR_BIT));
    fprintf(stdout, "Size of array: %.2f Kb\n", size_kb);
    fprintf(stdout, "Size of array: %.2f Mb\n", size_mb);
#endif
  }

  uint64_t N = (size_in_bytes + sizeof(uint64_t) - 1) / sizeof(uint64_t);

#ifdef DEBUG
  if(verbose)
  {
    fprintf(stdout, "N = %lu\n", N);
  }
#endif

  int coreId = 0;

  while( coreId < numcores)
  {

    if(coreId < 0 || coreId >= numcores)
    {
      fprintf(stdout, "FATAL ERROR! Invalid core id\n");
      exit(EXIT_FAILURE);
    }

    if(verbose)
    {
      fprintf(stdout, "Target core %d\n", coreId);
    }
    /* Set affinity mask to include CPUs coreId */

    CPU_ZERO(&cpuset);
    CPU_SET(coreId, &cpuset);

    s = pthread_setaffinity_np(thread, sizeof(cpu_set_t), &cpuset);
    if (s != 0)
      handle_error_en(s, "pthread_setaffinity_np");

    /* Check the actual affinity mask assigned to the thread */

    s = pthread_getaffinity_np(thread, sizeof(cpu_set_t), &cpuset);
    if (s != 0)
      handle_error_en(s, "pthread_getaffinity_np");

    for (j = 0; j < CPU_SETSIZE; j++)
    {
      if (CPU_ISSET(j, &cpuset))
      {
        cpu = j;
        break;
      }
    }

    if(j == CPU_SETSIZE)
    {
      fprintf(stderr, "FATAL ERROR! Don't know on which core the thread is placed\n");
      exit(EXIT_FAILURE);
    }

    int cur_numanode = numa_node_of_cpu(cpu);
    if(verbose)
    {
      fprintf(stdout, "Running on CPU %d of %d\n", cpu, numcores);
      fprintf(stdout, "Running on NUMA %d of %d\n", cur_numanode, numanodes);
    }

    for(int deviceId = 0; deviceId < gpucount; ++deviceId)
    {
      hipSetDevice(deviceId);
      if(verbose)
      {
        fprintf(stdout, "Set Device to %d\n", deviceId);
      }
      tgpu[coreId * gpucount + deviceId] = deviceId;

      hipStream_t stream;
      hipStreamCreate(&stream);

      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      uint64_t *A;
      hipHostMalloc(&A, N * sizeof(uint64_t), hipHostMallocDefault);

      for(int i = 0 ; i < N; ++i)
      {
        A[i] = i;
      }

      int allocnumaid = -1;
      get_mempolicy(&allocnumaid, NULL, 0, (void*)A, MPOL_F_NODE | MPOL_F_ADDR);
      if(allocnumaid != cur_numanode)
      {
        fprintf(stderr, "ERROR: bad NUMA allocation\n");
        hipHostFree(A);
        free(tgpu);
        free(HtD);
        free(DtH);
        free(HtD_gbs);
        free(DtH_gbs);
        exit(EXIT_FAILURE);
      }

      uint64_t *d_A;
      hipMalloc(&d_A, N * sizeof(uint64_t));

      duration = 0.;
      double throughput = 0.;
      double t0 = 0., t1 = 0.;
      hipDeviceSynchronize();
      for(int k = 0; k < nb_test; ++k)
      {
        hipStreamSynchronize(stream);

	t0 = get_elapsedtime();
        hipMemcpyAsync(d_A, A, N * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
	hipStreamSynchronize(stream);
	t1 = get_elapsedtime();

	if(k == 0) { continue; }
        duration += (t1 - t0);

#ifdef DEBUG
        get_mempolicy(&allocnumaid, NULL, 0, (void*)A, MPOL_F_NODE | MPOL_F_ADDR);
        if(allocnumaid != cur_numanode)
        {
          fprintf(stderr, "FATAL ERROR!!\n");
          exit(-1);
        }
#endif
      }

      duration /= nb_test-1;
      throughput = size_in_mbytes / (duration * 1000);
      if(verbose)
      {
        fprintf(stdout, "Performance results: \n");
        fprintf(stdout, "HostToDevice>  Time: %lf s\n", duration);
        fprintf(stdout, "HostToDevice>  Throughput: %.2lf GB/s\n", throughput);
      }
      HtD[coreId * gpucount + deviceId] = duration;
      HtD_gbs[coreId * gpucount + deviceId] = throughput;

      duration = 0.;
      t0 = t1 = 0.;
      hipDeviceSynchronize();
      for(int k = 0; k < nb_test; ++k)
      {
        hipStreamSynchronize(stream);

	t0 = get_elapsedtime();
        hipMemcpyAsync(A, d_A, N * sizeof(uint64_t), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	t1 = get_elapsedtime();

	if(k == 0) { continue; }
        duration += (t1 - t0);
      }

      duration /= nb_test-1;
      throughput = size_in_mbytes / (duration * 1000);
      if(verbose)
      {
        fprintf(stdout, "DeviceToHost>  Time: %lf s\n", duration);
        fprintf(stdout, "DeviceToHost>  Throughput: %.2lf GB/s\n\n", throughput);
      }
      DtH[coreId * gpucount + deviceId] = duration;
      DtH_gbs[coreId * gpucount + deviceId] = throughput;

      hipFree(d_A);
      hipHostFree(A);
      //coreId += numcores / numanodes;
    }
    coreId++;
  }

  char buff_memcpyasync_time[100];
  snprintf(buff_memcpyasync_time, 100, "%lu-MB_numa_memcpyasync_time.csv", size_in_mbytes);
  FILE * outputFile;
  outputFile = fopen( buff_memcpyasync_time, "w+" );
  if (outputFile == NULL)
  {
    printf( "Cannot open file %s\n", buff_memcpyasync_time );
    exit(EXIT_FAILURE);
  }

  fprintf(outputFile, "core\tgpu\tHostToDevice\tDeviceToHost\n");
  for(int i = 0; i < numcores; ++i)
  {
    for(int d = 0; d < gpucount; ++d)
    {
      fprintf(outputFile, "%d\t%d\t%lf\t%lf\n", i, tgpu[i * gpucount + d], HtD[i * gpucount + d], DtH[i * gpucount + d]);
    }
  }

  fclose(outputFile);

  char buff_memcpyasync_gbs[100];
  snprintf(buff_memcpyasync_gbs, 100, "%lu-MB_numa_memcpyasync_gbs.csv", size_in_mbytes);
  outputFile = fopen( buff_memcpyasync_gbs, "w+" );
  if (outputFile == NULL)
  {
    printf( "Cannot open file %s\n", buff_memcpyasync_gbs );
    exit(EXIT_FAILURE);
  }

  fprintf(outputFile, "core\tgpu\tHostToDevice\tDeviceToHost\n");
  for(int i = 0; i < numcores; ++i)
  {
    for(int d = 0; d < gpucount; ++d)
    {
      fprintf(outputFile, "%d\t%d\t%lf\t%lf\n", i, tgpu[i * gpucount + d], HtD_gbs[i * gpucount + d], DtH_gbs[i * gpucount + d]);
    }
  }

  fclose(outputFile);

  fprintf(stdout, "Results saved in:\n\tGB/s: %s\n", buff_memcpyasync_gbs);
  fprintf(stdout, "\tTime: %s\n", buff_memcpyasync_time);

  free(tgpu);
  free(HtD);
  free(DtH);
  free(HtD_gbs);
  free(DtH_gbs);

  exit(EXIT_SUCCESS);
}
